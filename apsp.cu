#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define B 64
#define threadNum 32

const int INF = 1000000000;
void input(char *inFileName);
void output(char *outFileName);

void block_FW();
int ceil(int a, int b);
__global__ void phase1(int* dist, int Round, int n, size_t pitch);
__global__ void phase2(int* dist, int Round, int n, size_t pitch);
__global__ void phase3(int* dist, int Round, int n, size_t pitch);

int n, m;	
int *Dist = NULL;
int *device_Dist = NULL;
size_t pitch;

int main(int argc, char* argv[]) {
	input(argv[1]);
	block_FW();
	output(argv[2]);
	hipHostFree(Dist);
	return 0;
}

void input(char* infile) {
    FILE* file = fopen(infile, "rb");
	fread(&n, sizeof(int), 1, file);
	fread(&m, sizeof(int), 1, file);
	hipHostMalloc(&Dist, (size_t)n*n*sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < n; ++ i) {
        for (int j = 0; j < n; ++ j) {
			Dist[i*n+j] = (i==j) ? 0 : INF;
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++ i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*n+pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char *outFileName) {
	FILE *outfile = fopen(outFileName, "wb");
	fwrite(Dist, sizeof(int), n*n, outfile);	
    fclose(outfile);
}

int ceil(int a, int b) {
	return (a + b - 1) / b;
}

void block_FW() {
	unsigned int round = ceil(n, B);
	dim3 block_p1 = {1, 1};
	dim3 block_p2 = {2, round};
	dim3 block_p3 = {round, round};
	dim3 threads = {threadNum, threadNum};
	hipMallocPitch(&device_Dist, &pitch, (size_t)n*sizeof(int), (size_t)n);
	hipMemcpy2D(device_Dist, pitch, Dist, (size_t)n*sizeof(int), (size_t)n*sizeof(int), (size_t)n, hipMemcpyHostToDevice);
	for (unsigned int r = 0; r < round; ++r) {
		phase1<<<block_p1, threads>>>(device_Dist, r, n, pitch);
		phase2<<<block_p2, threads>>>(device_Dist, r, n, pitch);
		phase3<<<block_p3, threads>>>(device_Dist, r, n, pitch);
	}
	hipMemcpy2D(Dist, (size_t)n*sizeof(int), device_Dist, pitch, (size_t)n*sizeof(int), (size_t)n, hipMemcpyDeviceToHost);
	hipFree(device_Dist);
}

__global__ void phase1(int* dist, int Round, int n, size_t pitch){

	int base = Round*B;
	int shift = B/threadNum;
	int i_st = base + threadIdx.x*shift, i_ed = i_st + shift;
	int j_st = base + threadIdx.y*shift, j_ed = j_st + shift;

	if(i_ed > n){
		i_ed = n;
	}
	if(j_ed > n){
		j_ed = n;
	}

	__shared__ int sm[B][B];

	#pragma unroll
	for(int i=i_st ; i<i_ed ; ++i){
		#pragma unroll
		for(int j=j_st ; j<j_ed ; ++j){
			int *dij = (int*)((char*)dist+pitch*i)+j;
			sm[i-base][j-base] = *dij;
		}
	}
	__syncthreads();

	int len = ((Round+1)*B < n) ? B : n - (Round)*B;

	#pragma unroll
	for (int k = 0; k < len; ++k) {
		#pragma unroll
		for(int i = i_st; i<i_ed ; ++i){
			#pragma unroll
			for(int j = j_st ; j<j_ed ; ++j){
				int relax = sm[i-base][k] + sm[k][j-base];
				if(relax < sm[i-base][j-base]){
					sm[i-base][j-base] = relax;
				}
			}
		}
		__syncthreads();
	}

	#pragma unroll
	for(int i=i_st ; i<i_ed ; ++i){
		#pragma unroll
		for(int j=j_st ; j<j_ed ; ++j){
			int *dij = (int*)((char*)dist+pitch*i)+j;
			*dij = sm[i-base][j-base];
		}
	}
}

__global__ void phase2(int* dist, int Round, int n, size_t pitch){
	if(blockIdx.y==Round)
		return;

	__shared__ int sm[2][B][B];
	
	int base_i = (1-blockIdx.x)*Round*B + blockIdx.x*blockIdx.y*B;
	int base_j = blockIdx.x*Round*B + (1-blockIdx.x)*blockIdx.y*B;
	int shift = B/threadNum;
	int i_st = base_i + threadIdx.x*shift, i_ed = i_st + shift; 
	int j_st = base_j + threadIdx.y*shift, j_ed = j_st + shift;	

	#pragma unroll
	for(int i=i_st ; i<i_ed ; ++i){
		#pragma unroll
		for(int j=j_st ; j<j_ed ; ++j){
			if(i<n && j<n){
				int *dij = (int*)((char*)dist+pitch*i)+j;
				sm[0][i-base_i][j-base_j] = *dij;
			}
			if(Round*B+(i-base_i)<n && Round*B+(j-base_j)<n){
				int *dkk = (int*)((char*)dist+pitch*(Round*B+(i-base_i))) + Round*B+(j-base_j);
				sm[1][i-base_i][j-base_j] = *dkk;
			}
		}
	}
	__syncthreads();

	if(i_ed > n){
		i_ed = n;
	}
	if(j_ed > n){
		j_ed = n;
	}
	int len = ((Round+1)*B < n) ? B : n - (Round)*B;
	int i_offset = i_st-base_i, i_len = i_ed - i_st;
	int j_offset = j_st-base_j, j_len = j_ed - j_st;
	
	#pragma unroll
	for(int i=i_offset ; i<i_offset+i_len ; ++i){
		#pragma unroll
		for(int j=j_offset ; j<j_offset+j_len ; ++j){
			#pragma unroll
			for (int k = 0; k < len; ++k) {
				int relax = sm[1-blockIdx.x][i][k] + sm[blockIdx.x][k][j];
				if(relax < sm[0][i][j]){
					sm[0][i][j] = relax;
				}
			}
			int *dij = (int*)((char*)dist+pitch*(base_i+i))+base_j+j;
			*dij = sm[0][i][j];
		}
	}
}

__global__ void phase3(int* dist, int Round, int n, size_t pitch){
	if(blockIdx.x==Round || blockIdx.y==Round)
		return;

	__shared__ int sm[2][B][B];
	
	int base_i = blockIdx.x*B;
	int base_j = blockIdx.y*B;
	int shift = B/threadNum;
	int i_st = base_i + threadIdx.x*shift, i_ed = i_st + shift;
	int j_st = base_j + threadIdx.y*shift, j_ed = j_st + shift;

	#pragma unroll
	for(int i=i_st ; i<i_ed ; ++i){
		#pragma unroll
		for(int j=j_st ; j<j_ed ; ++j){
			if(i<n && Round*B+(j-base_j)<n){
				int *dik = (int*)((char*)dist+pitch*i)+Round*B+(j-base_j);
				sm[0][j-base_j][i-base_i] = *dik;
			}
			if(Round*B+(i-base_i)<n && j<n){
				int *dkj = (int*)((char*)dist+pitch*(Round*B+(i-base_i)))+j;
				sm[1][i-base_i][j-base_j] = *dkj;
			}
		}
	}
	__syncthreads();

	if(i_ed > n){
		i_ed = n;
	}
	if(j_ed > n){
		j_ed = n;
	}
	int len = ((Round+1)*B < n) ? B : n - (Round)*B;
	int i_offset = i_st-base_i, i_len = i_ed - i_st;
	int j_offset = j_st-base_j, j_len = j_ed - j_st;

	#pragma unroll
	for(int i = 0 ; i < i_len ; ++i){
		#pragma unroll
		for(int j= 0 ; j < j_len ; ++j){
			int *dij = (int*)((char*)dist+pitch*(i_st+i))+j_st+j;
			int ans = *dij;
			#pragma unroll
			for (int k = 0; k < len; ++k) {
				int relax = sm[0][k][i_offset+i] + sm[1][k][j_offset+j];
				if(relax < ans){
					ans = relax;
				}
			}
			*dij = ans;
		}
	}
}